/*

This program will numerically compute the integral of

                  4/(1+x*x) 
				  
from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>

static long num_steps = 100000000;
double step;
int num_blocks = 1;

__global__ 
void step_func(double* d_result, double step_c, long num_steps) {
    int i  = blockIdx.x;  
    double block_result = 0.0;
    double x; 
    int block_iter_size = (int) ceil((double) num_steps / (double) gridDim.x);
    for (int j = 0; j < block_iter_size ; j++){
	if(i * block_iter_size + j >= num_steps)
	    break;
	x = (i * block_iter_size + j - 0.5) * step_c;
	block_result += 4.0/(1.0+x*x); 	
    }
    __syncthreads();
    printf("block result for %d is %lf, and the size is %ld \n", i, block_result, (int)ceil((double) num_steps / (double) gridDim.x));
    atomicAdd(d_result, block_result); 
}
/*	  for (i=1;i<= num_steps; i++){
		  x = (i-0.5)*step;
		  sum = sum + 4.0/(1.0+x*x);
	  }

	  pi = step * sum;
*/

int main (int argc, char** argv)
{
    
      // Read command line arguments.
      for ( int i = 0; i < argc; i++ ) {
        if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
            num_steps = atol( argv[ ++i ] );
            printf( "  User num_steps is %ld\n", num_steps );
        }
	if ( ( strcmp( argv[ i ], "-B" ) == 0 ) || ( strcmp( argv[ i ], "-num_blocks" ) == 0 ) ) {
            num_blocks = atol( argv[ ++i ] );
            printf( "  User num_block is %ld\n", num_blocks);
        } else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
            printf( "  Pi Options:\n" );
            printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
            printf( "  -help (-h):            print this message\n\n" );
            exit( 1 );
        }
      }
      
	  double pi;

	  double *h_result;
	  
      step = 1.0/(double) num_steps;
      if(num_blocks > num_steps){
	num_blocks = num_steps; 
	printf("num_blocks specified are greater than num_steps, num_blocks are set to num_steps.");
      }
      // Timer products.
      struct timeval begin, end;

      gettimeofday( &begin, NULL );

// ***************** START of the part to parallelize with CUDA.
// we start with a block of 1 thread, which means we will have the number of blocks equals the number of steps.
// but there is a limit to the number of blocks that can be used in a grid. on x 2^31 -1  
    

// init device variables 

double *d_block, *d_result; 
hipMalloc((void **) &d_result, sizeof(double));
hipMalloc((void **) &d_block, num_steps * sizeof(double));

// cpy host variables into the device variables 
//cudaMemcpy(d_result, h_result, sizeof(double), cudaMemcpyHostToDevice); 
// cpy the device variable to the host variable 

step_func<<<num_blocks,1>>>(d_result, step, num_steps); 
hipMemcpy(h_result, d_result,  sizeof(double), hipMemcpyDeviceToHost); 

// free the device variables 
pi = step * (*h_result);
// ***************** END of the part to parallelize with CUDA.
      
hipFree(d_result);
      gettimeofday( &end, NULL );

      // Calculate time.
      double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
                1.0e-6 * ( end.tv_usec - begin.tv_usec );
                
      printf("\n pi with %ld steps, step= %lf, result = %lf , is %lf in %lf seconds\n ",num_steps,step,*h_result,pi,time);
}

